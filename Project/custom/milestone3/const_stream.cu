#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#define TILE_WIDTH 12

__constant__ float ConstMask[8192];

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;


#define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
#define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
#define mask_4d(i3, i2, i1, i0) ConstMask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    int W_grid = (Width_out - 1)/ TILE_WIDTH + 1;
    // Insert your GPU convolution kernel code here
    int h = (blockIdx.z / W_grid) * TILE_WIDTH + threadIdx.y;
    int w = (blockIdx.z % W_grid) * TILE_WIDTH + threadIdx.x;
    if ((h < Height_out) && (w < Width_out)){
        float res = 0;
        for (int c=0; c<Channel; c++){
            for (int p=0; p<K; p++){
                for (int q=0; q<K; q++){
                    res += in_4d(blockIdx.x, c, h+p, w+q) * mask_4d(blockIdx.y, c, p, q);
                }
            }    
        }
        out_4d(blockIdx.x, blockIdx.y, h ,w) = res;
    }



#undef out_4d
#undef in_4d
#undef mask_4d
}

__host__ void GPUInterface::conv_forward_gpu_prolog(float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    int SegSize = 10;
    hipMalloc((void **) device_output_ptr, (Batch * Map_out * Height_out * Width_out) * sizeof(float));
    hipMalloc((void **) device_input_ptr, (Batch * Channel * Height * Width) * sizeof(float));
    
    int inputSize = Height * Width * Channel;
    int outputSize = Height_out * Width_out * Map_out;
    hipMemcpyToSymbol(HIP_SYMBOL(ConstMask), host_mask, (Map_out * Channel * K * K) * sizeof(float));

    //dim3 dimGrid(Seg, M, ceil((float)(H - K + 1)/TILE_WIDTH)*ceil((float)(W - K + 1)/TILE_WIDTH));
    dim3 dimGrid(SegSize, Map_out, ceil((float)Height_out / TILE_WIDTH) * ceil((float)Width_out / TILE_WIDTH));
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    hipStream_t stream0, stream1, stream2, stream3, stream4;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);
    
    for (int i = 0; i < Batch; i += 5*SegSize){        
        hipMemcpyAsync(*device_input_ptr + i * inputSize, host_input + i * inputSize, SegSize * inputSize * sizeof(float), hipMemcpyHostToDevice,stream0);
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream0>>>(*device_output_ptr + i * outputSize, *device_input_ptr + i * inputSize, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        hipMemcpyAsync(host_output + i * outputSize, *device_output_ptr + i * outputSize, SegSize * outputSize * sizeof(float), hipMemcpyDeviceToHost, stream0);
        
        hipMemcpyAsync(*device_input_ptr + (i + 1 * SegSize) * inputSize, host_input + (i + 1 * SegSize) * inputSize, SegSize * inputSize * sizeof(float), hipMemcpyHostToDevice,stream1);
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream1>>>(*device_output_ptr + (i + 1 * SegSize) * outputSize, *device_input_ptr + (i + 1 * SegSize) * inputSize, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        hipMemcpyAsync(host_output + (i + 1 * SegSize) * outputSize, *device_output_ptr + (i + 1 * SegSize) * outputSize, SegSize * outputSize * sizeof(float), hipMemcpyDeviceToHost, stream1);
        
        hipMemcpyAsync(*device_input_ptr + (i + 2 * SegSize) * inputSize, host_input + (i + 2 * SegSize) * inputSize, SegSize * inputSize * sizeof(float), hipMemcpyHostToDevice,stream2);
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream2>>>(*device_output_ptr + (i + 2 * SegSize) * outputSize, *device_input_ptr + (i + 2 * SegSize) * inputSize, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        hipMemcpyAsync(host_output + (i + 2 * SegSize) * outputSize, *device_output_ptr + (i + 2 * SegSize) * outputSize, SegSize * outputSize * sizeof(float), hipMemcpyDeviceToHost, stream2);        
        
        hipMemcpyAsync(*device_input_ptr + (i + 3 * SegSize) * inputSize, host_input + (i + 3 * SegSize) * inputSize, SegSize * inputSize * sizeof(float), hipMemcpyHostToDevice,stream3);
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream3>>>(*device_output_ptr + (i + 3 * SegSize) * outputSize, *device_input_ptr + (i + 3 * SegSize) * inputSize, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        hipMemcpyAsync(host_output + (i + 3 * SegSize) * outputSize, *device_output_ptr + (i + 3 * SegSize) * outputSize, SegSize * outputSize * sizeof(float), hipMemcpyDeviceToHost, stream3);        
        
        hipMemcpyAsync(*device_input_ptr + (i + 4 * SegSize) * inputSize, host_input + (i + 4 * SegSize) * inputSize, SegSize * inputSize * sizeof(float), hipMemcpyHostToDevice,stream4);
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream4>>>(*device_output_ptr + (i + 4 * SegSize) * outputSize, *device_input_ptr + (i + 4 * SegSize) * inputSize, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        hipMemcpyAsync(host_output + (i + 4 * SegSize) * outputSize, *device_output_ptr + (i + 4 * SegSize) * outputSize, SegSize * outputSize * sizeof(float), hipMemcpyDeviceToHost, stream4);
    }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // do nothing
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // cuda free
    hipFree(device_input); 
    hipFree(device_output);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
