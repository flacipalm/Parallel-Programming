#include "hip/hip_runtime.h"

#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)



__global__  void Cal_cross_sum(){}
  __shared__ float subTile1[tile_width][tile_width]
  __shared__ float subTile2[tile_width][tile_width]
 //@@ Insert code to implement matrix multiplication 
  const int TILE_WIDTH = 5;
  __shared__ float subTileM[TILE_WIDTH][TILE_WIDTH];
  __shared__ float subTileN[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  
  int width = numAColumns;
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  float Pvalue = 0;

  for (int q = 0; q < ((width - 1) / TILE_WIDTH + 1); ++q){
    if ((Row < numARows) && ((q * TILE_WIDTH + tx) < width)){
      subTileM[ty][tx] = A[Row * width + q * TILE_WIDTH + tx];
    } else{
      subTileM[ty][tx] = 0;
    }

    if ((q * TILE_WIDTH + ty < width) && (Col < numBColumns)){
      subTileN[ty][tx] = B[(q * TILE_WIDTH + ty) * numBColumns + Col];
    } else{
      subTileN[ty][tx] = 0;
    }

    __syncthreads();
    if ((Row < numCRows) && (Col < numCColumns)){
      for (int k = 0; k < TILE_WIDTH; ++k){
        Pvalue += subTileM[ty][k] * subTileN[k][tx];
      }
    }
    __syncthreads();
  }
  if ((Row < numARows) && (Col < numCColumns)){
    C[Row * numCColumns + Col] = Pvalue;
  }

__host__ void CrossSum(int *Data, int *OutData, int size){
  hipMalloc((void **) &deviceA, sizeA);
  hipMalloc((void **) &deviceB, sizeB);
  hipMalloc((void **) &deviceC, sizeC);


}






// Compute C = A * B
__host__ void CrossSum(int *Data, int *OutData, int size){

}
 
}


__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
  //@@ Insert code to implement matrix multiplication 
  const int TILE_WIDTH = 2;
  __shared__ float subTileM[TILE_WIDTH][TILE_WIDTH];
  __shared__ float subTileN[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  
  int width = numAColumns;
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  float Pvalue = 0;

  for (int q = 0; q < ((width - 1) / TILE_WIDTH + 1); ++q){
    if ((Row < numARows) && ((q * TILE_WIDTH + tx) < width)){
      subTileM[ty][tx] = A[Row * width + q * TILE_WIDTH + tx];
    } else{
      subTileM[ty][tx] = 0;
    }

    if ((q * TILE_WIDTH + ty < width) && (Col < numBColumns)){
      subTileN[ty][tx] = B[(q * TILE_WIDTH + ty) * numBColumns + Col];
    } else{
      subTileN[ty][tx] = 0;
    }

    __syncthreads();
    if ((Row < numCRows) && (Col < numCColumns)){
      for (int k = 0; k < TILE_WIDTH; ++k){
        Pvalue += subTileM[ty][k] * subTileN[k][tx];
      }
    }
    __syncthreads();
  }
  if ((Row < numARows) && (Col < numCColumns)){
    C[Row * numCColumns + Col] = Pvalue;
  }
}



int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  wbTime_stop(Generic, "Importing data and creating memory on host");
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  int sizeA = numARows * numAColumns * sizeof(float);
  int sizeB = numBRows * numBColumns * sizeof(float);
  int sizeC = numCRows * numCColumns * sizeof(float);

  hipMalloc((void **) &deviceA, sizeA);
  hipMalloc((void **) &deviceB, sizeB);
  hipMalloc((void **) &deviceC, sizeC);

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, sizeB, hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  int TILE_WIDTH = 2;
  dim3 DimGrid(ceil((1.0 * numCColumns) / TILE_WIDTH), ceil((1.0 * numCRows) / TILE_WIDTH), 1); // (x, y, z)
  dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiply<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
